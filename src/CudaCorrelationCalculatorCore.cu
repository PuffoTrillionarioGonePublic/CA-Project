#include "hip/hip_runtime.h"
// Headers included by Visual Studio, unnecessary
#include <hip/hip_runtime.h>
#include <>

#include "../modules/CPP-csv-parser/csv.hh"
#include "../modules/CPP-math-utils/convertions.hh"
#include <math.h>

#include <stdio.h>
#include <istream>
#include <fstream>
#include <string>
#include <utility>
#include <optional>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

using T = double;

// Macro can be used anywhere, regardeless of CUDA support for C++ code
#define COUPLE_NUMBER(n) ((n-1)*(n)/2)

// MIN macro
#define MIN(x,y) ((x) < (y) ? (x) : (y))


struct Pair {
    int first = 0, second = 0;
};

template <typename T>
struct PCC_Partial {
    long long count{};
    T sum_1{};
    T sum_2{};
    T sum_1_squared{};
    T sum_2_squared{};
    T sum_prod{};
};


// Cuda seems not supporting classes, this function was the core
// of the math::sets::couple class so I extracted it to
//  https://github.com/Tredici/CPP-math-utils/blob/b3b3f844d51b1014e20d329009461b8ac74ef21d/couple.hh#L18
__device__ Pair pair(int n, int i) {
    // candidate supposing al pairs are ok
    Pair p{ i / n, i % n };

    // first column and no overflow?
    if (p.first == 0 && p.second + 1 < n) {
        p.second += 1;
        return p;
    }
    else if (p.first == 0 && p.second + 1 == n) {
        p.first = 1; p.second = 2;
        return p;
    }

    // reduce problem with recursion
    // [0,1] for new base will be
    // translated to [p[0], p[0]+1]
    Pair base{ p.first, p.first };
    // all points in the triangle
    // marked by [p[0], p[0]]
    // must be ignored, others must be
    // counted
    auto remaining = i - ((n - 1) * p.first - (p.first - 1) * p.first / 2);
    auto p2 = ::pair(n - p.first, remaining);
    p2.first += base.first, p2.second += base.second;
    return p2;
}

// Implement the same op without using recursion
__device__  Pair fast_pair(const int n, const int p)
{
    // closed form formula from: https://stackoverflow.com/questions/21331385/indexing-the-unordered-pairs-of-a-set
    const int to_square = 2 * n - 1;
    const int square = to_square * to_square;
    // floorf
    //  https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE
    const int x = floorf((to_square - sqrt(float(square - 8 * p))) / 2);
    const int y = p - (2 * n - x - 3) * x / 2 + 1;
    return Pair{ x, y };
}


__device__ Pair inc(int n, Pair couple) {
    if (++couple.second == n) {
        couple.first += 1;
        couple.second = couple.first + 1;
        return couple;
    }
    return couple;
}


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Return a vector that will hold partial results
template <typename T>
auto allocate_partial_container(int ts_count) {
    thrust::device_vector<PCC_Partial<T>> ans(COUPLE_NUMBER(ts_count));
    hipMemset(thrust::raw_pointer_cast(&ans[0]), 0, sizeof(PCC_Partial<T>) * ans.size());
    return ans;
}

template <typename T>
auto allocate_result_container(int ts_count) {
    thrust::device_vector<T> ans(COUPLE_NUMBER(ts_count));
    hipMemset(thrust::raw_pointer_cast(&ans[0]), 0, sizeof(T) * ans.size());
    return ans;
}

template <typename T>
__device__ void operator+=(PCC_Partial<T>& p1, const PCC_Partial<T>& p2) {
    p1.count += p2.count;
    p1.sum_1 += p2.sum_1;
    p1.sum_2 += p2.sum_2;
    p1.sum_1_squared += p2.sum_1_squared;
    p1.sum_2_squared += p2.sum_2_squared;
    p1.sum_prod += p2.sum_prod;
}

// comput partial pcc on two time series
template <typename T>
__device__ void calculate_pcc(PCC_Partial<T>* partial, const T* v1, const T* v2, int length) {
    PCC_Partial<T> ans{};
    for (int i{}; i != length; ++i) {
        const auto v_1 = v1[i];
        const auto v_2 = v2[i];
        ans.sum_1 += v_1;
        ans.sum_2 += v_2;
        ans.sum_1_squared += v_1 * v_1;
        ans.sum_2_squared += v_2 * v_2;
        ans.sum_prod += v_1 * v_2;
    }
    ans.count = length;
    *partial += ans;
}

template <typename T>
__global__ void evaluate(PCC_Partial<T>* partial, T** chunk, int length, int lines) {
    int limit = COUPLE_NUMBER(length);
    // to linearize blocks
    auto id = threadIdx.x + blockDim.x * blockIdx.x;
    auto poolsz = (gridDim.x ? gridDim.x : 1) * blockDim.x;
    // columns per thread
    auto cpt = limit / poolsz + (limit % poolsz != 0);
    // more thread than items? Might happet if columns are too few
    if (cpt == 0) {
        if (id < limit) {
            auto i = id;
            //auto couple = ::pair(length, i);
            auto couple = ::fast_pair(length, i);
            calculate_pcc(&partial[i], chunk[couple.first], chunk[couple.second], lines);
        }
    }
    // else at least one columnt per thread
    else {
        // for each pair assigned to this thread
        auto beginning = cpt * id;
        auto end = MIN(beginning + cpt, limit);
        //auto couple = ::pair(length, beginning);
        auto couple = ::fast_pair(length, beginning);
        while (beginning < end) {
            calculate_pcc(&partial[beginning], chunk[couple.first], chunk[couple.second], lines);
            // next pair
            couple = ::inc(length, couple);
            ++beginning;
        }
    }
}

template <typename T>
__device__ T compute(const PCC_Partial<T>& pcc) {
    if (pcc.count == 0) {
        return 0;
    }
    const auto num = (pcc.sum_prod - (pcc.sum_1 * pcc.sum_2) / pcc.count);
    const auto den = (pcc.sum_1_squared - (pcc.sum_1 * pcc.sum_1 / pcc.count)) * (pcc.sum_2_squared - (pcc.sum_2 * pcc.sum_2 / pcc.count));
    // check for div by 0
    return den ? num / sqrt(den) : 0;
}

// calculate results element by element
template <typename T>
__global__ void compute_results(int n, T* res, const PCC_Partial<T>* partials) {
    int limit = COUPLE_NUMBER(n);
    // pairs of columns per thread
    auto cpt = limit / blockDim.x + (limit % blockDim.x != 0);
    // more thread than items? Might happet if columns are too few
    if (cpt == 0) {
        auto i = threadIdx.x;
        if (i < limit) {
            res[i] = compute(partials[i]);
        }
    }
    // else at least one columnt per thread
    else {
        // for each pair assigned to this thread
        const auto beginning = cpt * threadIdx.x;
        const auto end = MIN(beginning + cpt, limit);
        // compute final result
        for (int i = beginning; i < end; ++i) {
            res[i] = compute(partials[i]);
        }
    }
}

void print_results(const int n, const thrust::host_vector<T>& hres) {
    int p = 0;
    for (int i{}; i != n-1; ++i) {
        for (int j{ i + 1 }; j != n; ++j) {
            std::cout << "(" << i << "," << j << ") " << hres[p++] << '\n';
        }
    }
}

template <typename T>
std::vector<thrust::device_vector<T>>& get_chunk(csv::reader& r, std::size_t line_count) {
    static auto column_count = r.column_count();
    static std::vector<thrust::host_vector<T>> tmp(column_count);
    // many columns as tmp
    static auto ans = std::vector<thrust::device_vector<T>>(column_count);
    static bool first = true;
    if (first) {
        // execute once
        first = false;
        for (auto& v : tmp) {
            // allocate space for line count rows on CPU memory
            v.resize(line_count);
        }
        // allocate space for line count rows on GPU memory
        auto& ans_v = ans;
        for (auto& v : ans_v) {
            // allocate space for line count rows on CPU memory
            v.resize(line_count);
        }
    }
    auto inserted_rows = 0;
    while (r.can_read() && inserted_rows != line_count) {
        auto line = r.getline();
        for (size_t i = 0; i < column_count; ++i) {
            tmp[i][inserted_rows] = math::convertions::ston<T>(line.data()[i]);
        }
        ++inserted_rows;
    }
    if (inserted_rows == 0) {
        ans.clear();
        return ans;
    }
    // if found less rows than expeceted, shrink vector
    else if (inserted_rows != line_count) {
        for (auto& v : tmp) {
            v.resize(inserted_rows);
        }
    }
    // copy data to GPU
    {
        auto& ans_v = ans;
        for (decltype(column_count) i{}; i != column_count; ++i) {
            ans_v[i] = tmp[i];
        }
    }
    return ans;
}

template <typename T>
void print(const PCC_Partial<T>& pcc) {
    std::cout
        << "count:         " << pcc.count << '\n'
        << "sum_1:         " << pcc.sum_1 << '\n'
        << "sum_1_squared: " << pcc.sum_1_squared << '\n'
        << "sum_2:         " << pcc.sum_2 << '\n'
        << "sum_2_squared: " << pcc.sum_2_squared << '\n'
        << "sum_prod:      " << pcc.sum_prod << '\n';
}

template <typename T>
thrust::host_vector<T> cuda_evaluate(csv::reader& r, std::size_t chunk_size)
{
    auto ts_count = static_cast<int>(r.column_count());

    // since this allocation the map containing
    // the results will never change its size,
    // so it is safe to access distinct elements
    // from different thread because no ops will
    // be performed on its structure
    auto partial = allocate_partial_container<T>(ts_count);

    while (true) {
        auto& chunk = get_chunk<T>(r, chunk_size);
        if (chunk.empty()) {
            break;
        }
        thrust::device_vector<T*> c;
        c.reserve(chunk.size());
        for (auto& v : chunk) {
            c.push_back(thrust::raw_pointer_cast(&v[0]));
        }
        evaluate<<<16,1024>>>(thrust::raw_pointer_cast(&partial[0]), thrust::raw_pointer_cast(&c[0]), ts_count, chunk[0].size());
    }
    //print(partial[0]);
    auto res = allocate_result_container<T>(ts_count);
    compute_results<<<16,1024>>>(ts_count, thrust::raw_pointer_cast(&res[0]), thrust::raw_pointer_cast(&partial[0]));
	thrust::host_vector<T> hres(res);
    return hres;
}


std::vector<double> cuda_evaluate_double(csv::reader& r, std::size_t chunk_size) {
	thrust::host_vector<double> hv = cuda_evaluate<double>(r, chunk_size);
	return std::vector<double>(hv.begin(), hv.end());
}

std::vector<float> cuda_evaluate_float(csv::reader& r, std::size_t chunk_size) {
	thrust::host_vector<float> hv = cuda_evaluate<double>(r, chunk_size);
	return std::vector<float>(hv.begin(), hv.end());
}
